#include <iostream>
#include "hip/hip_runtime.h"
#include <vector>

using namespace std;

__global__ void VecAdd(int* A, int* B, int* C) {
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

class CudaVecAdd {
public:
    bool operator()(vector<int> &a, vector<int> &b, vector<int> &c);
};

bool CudaVecAdd::operator()(vector<int>& a, vector<int>& b, vector<int>& c) {
    hipError_t err = hipSuccess;
    int *A, *B, *C;
    cout<<"size: "<<a.size()<<endl;
    err = hipMalloc<int>(&A, a.size() * sizeof(int));
    if (err != hipSuccess) {
        cerr<<"hipMalloc failed. err: "<<err<<endl;
        return false;
    }
    err = hipMalloc<int>(&B, a.size() * sizeof(int));
    if (err != hipSuccess) {
        cout<<"hipMalloc failed. err: "<<err<<endl;
        return false;
    }

    err = hipMalloc<int>(&C, a.size() * sizeof(int));
    if (err != hipSuccess) {
        cout<<"the hipMalloc on GPU is failed. err: "<<err<<endl;
        return false;
    }
    err  = hipMemcpy(A, &(a.front()), a.size() * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        cerr<<"cudaMmecpy failed. err: "<<err<<endl;
    }
    err = hipMemcpy(B, &(b.front()), sizeof(int) * b.size(), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        cerr<<"cudaMmecpy failed. err: "<<err<<endl;
    }
    VecAdd<<<1,a.size()>>>(A, B, C);
    err = hipMemcpy(&c.front(), C, a.size() * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        cerr<<"hipMemcpy device to host failed. err:"<<err<<endl;
    }
    hipFree(A);
    hipFree(B);
    hipFree(C);

    return true;
}

// 一个block 内存线程数据有限，如果数据量很大，需要用多个block来做。
int main() {
    CudaVecAdd cuda_va;
    vector<int> a;
    vector<int> b;
    vector<int> c;
    const int size = 1000;  

    a.resize(size, 0);
    b.resize(size, 0);
    c.resize(size, 0);

    for (int i = 0; i < size; ++i) {
        a[i] = i;
        b[i] = i;
    }
    cuda_va(a, b, c);
    for(int i = 0; i < 100; ++i) {
        cout<<" "<<c[i];
    }
    cout<<endl;

    return 0;
}
